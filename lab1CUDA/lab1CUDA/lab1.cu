#include "hip/hip_runtime.h"
#include<iostream>
#include<math.h>
#define degree 32
using namespace std;
__global__ void kernel(long long* a) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < degree) {
        a[idx] = pow(2, idx);
     };
}
int main() {
    int thread = 1024;
    long long b[degree];
    long long* dev;
    int bytes = degree * sizeof(long long);

    hipMalloc(&dev, bytes);

    hipMemcpy(dev, b, bytes, hipMemcpyHostToDevice);

    kernel << < 1, thread >> > (dev);

    hipMemcpy(b, dev, bytes, hipMemcpyDeviceToHost);

    hipFree(dev);

    for (int i = 0; i < degree; i++) printf("2^%d = %lld\n", i, b[i]);

    return 0;
}












