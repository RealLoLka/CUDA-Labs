#include "hip/hip_runtime.h"
#include<iostream>
#include<math.h>
#include <chrono>
#include <stdio.h>
#include <stdlib.h>
using namespace std;
using namespace chrono;

#define degree 50000



__global__ void kernel(long long* a) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < degree) {
        a[idx] = pow(2, idx);
     };
}

void tcpu(long long *a, int N) {
    int i = 0;
    for (i = 0; i < N; i++) {
        a[i] = pow(2, i);
        }
}



int main() {
    int thread = 1024;
    long long b[degree];
    long long b_cpu[degree];
    long long* dev;
    int bytes = degree * sizeof(long long);

    hipMalloc(&dev, bytes);

    hipMemcpy(dev, b, bytes, hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    kernel << < 50, thread >> > (dev);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(b, dev, bytes, hipMemcpyDeviceToHost);

    hipFree(dev);

    const auto before = system_clock::now();
    tcpu(b_cpu, degree);
    const duration<double> sec = system_clock::now() - before;

    for (int i = 0; i < degree; i++) printf("GPU result 2^%d = %lld\n CPU result 2^%d = %lld\n ", i, b[i], i, b_cpu[i]);


    printf("Time elapsed on GPU: %f ms\n",gpuTime);
    printf("Time elapsed on CPU: %f ms\n", sec.count());

    return 0;
}